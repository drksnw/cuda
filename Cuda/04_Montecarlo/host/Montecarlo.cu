#include "hip/hip_runtime.h"
#include "Montecarlo.h"

#include <iostream>
#include <hiprand/hiprand_kernel.h>

#include "Device.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void montecarlo(hiprandState* ptrTabDevGeneratorGM,int* ptrDevN0, float a, float b, float M, int nbFlechettes);
extern __global__ void setup_kernel_rand(hiprandState* tabGeneratorThread, int deviceId);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Constructeur			*|
 \*-------------------------------------*/

Montecarlo::Montecarlo(const Grid& grid, float a, float b, float M, int nbFlechettes) :
		a(a), b(b), M(M), nbFlechettes(nbFlechettes)
{

	this->dg = grid.dg;
	this->db = grid.db;

	this->sizeOctetTabGenerator = dg.x * dg.y * dg.z * db.x * db.y * db.z * sizeof(hiprandState); // octet
	this->sizeOctetN0 = sizeof(int);
	this->sizeSM = db.x * db.y * db.z * sizeof(int);

	// MM
	{
		// MM (malloc Device)
		{
			HANDLE_ERROR(hipMalloc(&ptrDevN0, sizeOctetN0));
			HANDLE_ERROR(hipMalloc(&ptrTabDevGeneratorGM, sizeOctetTabGenerator));
		}

		// MM (memset Device)
		{
			HANDLE_ERROR(hipMemset(ptrDevN0, 0, sizeOctetN0));
		}

		Device::lastCudaError("Montecarlo MM (end allocation)"); // temp debug
	}

	setup_kernel_rand<<<dg, db>>>(ptrTabDevGeneratorGM, Device::getDeviceId());
}

Montecarlo::~Montecarlo(void)
{
	//MM (device free)
	{
		HANDLE_ERROR(hipFree(ptrDevN0));
		HANDLE_ERROR(hipFree(ptrTabDevGeneratorGM));

		Device::lastCudaError("Montecarlo MM (end deallocation)"); // temp debug
	}
}

/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

float Montecarlo::getPi()
{
	return this->pi;
}

void Montecarlo::run()
{
	Device::lastCudaError("Montecarlo (before)"); // temp debug
	montecarlo<<<dg,db, sizeSM>>>(ptrTabDevGeneratorGM, ptrDevN0, a, b, M, nbFlechettes); // assynchrone
	Device::lastCudaError("Montecarlo (after)"); // temp debug

	Device::synchronize(); // Temp, only for printf in  GPU

	// MM (Device -> Host)
	{
		HANDLE_ERROR(hipMemcpy(&N0, ptrDevN0, sizeOctetN0, hipMemcpyDeviceToHost)); // barriere synchronisation implicite
	}

	float delta = fabsf(b - a);
	float rektArea = M * delta;
	float ratioFlechette = N0 / (float)nbFlechettes;
	pi = 2 * rektArea * ratioFlechette;
}

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
