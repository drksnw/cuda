#include "hip/hip_runtime.h"
#include "Device.h"
#include "Raytracing.h"

extern __global__ void raytracing(uchar4* ptrDevPixels, Sphere* ptrDevTabSphere, uint w, uint h, uint nbSphere, float t); //CUDA Kernel

Raytracing::Raytracing(const Grid& grid, uint w, uint h, float dt, int nbSpheres) :
		Animable_I<uchar4>(grid, w, h, "Raytracing_Cuda_RGBA_uchar4")
    {
    this->dt = dt;
    this->nbSpheres = nbSpheres;
    this->sizeOctet = nbSpheres * sizeof(Sphere);

    this->t = 0;

    SphereCreator sphereCreator(nbSpheres, w, h);
    Sphere* ptrTabSphere = sphereCreator.getTabSphere();

    //TODO: Transfer tab to GM
    Device::malloc(&ptrDevTabSphere, sizeOctet);
    Device::memclear(ptrDevTabSphere, sizeOctet);
    Device::memcpyHToD(ptrDevTabSphere, ptrTabSphere, sizeOctet);

    }

Raytracing::~Raytracing()
    {
    Device::free(ptrDevTabSphere);
    }

void Raytracing::animationStep()
    {
    t += dt;
    }

void Raytracing::process(uchar4* ptrDevPixels, uint w, uint h, const DomaineMath& domaineMath)
    {
    Device::lastCudaError("raytracing before");
    raytracing<<<dg, db>>>(ptrDevPixels, ptrDevTabSphere, w, h, this->nbSpheres, t);
    Device::lastCudaError("raytracing after");
    }
