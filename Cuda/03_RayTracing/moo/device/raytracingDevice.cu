#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "cudaTools.h"
#include "Device.h"

#include "IndiceTools_GPU.h"

#include "RaytracingMath.h"
using namespace gpu;

__global__ void raytracing(uchar4* ptrDevPixels, Sphere* ptrDevTabSphere, uint w, uint h, uint nbSphere, float t);

__global__ void raytracing(uchar4* ptrDevPixels, Sphere* ptrDevTabSphere, uint w, uint h, uint nbSphere, float t)
    {
    RaytracingMath raytracingMath(ptrDevTabSphere, nbSphere);
    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();
    const int WH = w * h;

    int s = TID;
    int i;
    int j;

    while(s < WH)
	{
	IndiceTools::toIJ(s, w, &i, &j);
	raytracingMath.colorXY(&ptrDevPixels[s], i, j, t);
	s += NB_THREAD;
	}
    }
