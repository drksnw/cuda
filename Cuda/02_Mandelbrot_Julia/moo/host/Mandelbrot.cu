#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>

#include "Device.h"
#include "Mandelbrot.h"
#include <assert.h>

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void mandelbrot(uchar4* ptrDevPixels,uint w, uint h,float t, DomaineMath domaineMath);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

Mandelbrot::Mandelbrot(const Grid& grid, uint w, uint h, float dt, const DomaineMath& domaineMath) :
	Animable_I<uchar4>(grid, w, h, "Mandelbrot_Cuda_RGBA_uchar4", domaineMath), variateurAnimation(Interval<float>(0, 120), dt)
    {
    // Inputs
    this->dt = dt;

    // Tools
    this->t = 0; // protected dans Animable
    }

Mandelbrot::~Mandelbrot()
    {
    // rien
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 * Call periodicly by the API
 *
 * Note : domaineMath pas use car pas zoomable
 */
void Mandelbrot::process(uchar4* ptrDevPixels, uint w, uint h, const DomaineMath& domaineMath)
    {
    Device::lastCudaError("rippling rgba uchar4 (before kernel)"); // facultatif, for debug only, remove for release

    mandelbrot<<<dg,db>>>(ptrDevPixels, w, h, t, domaineMath);

    Device::lastCudaError("rippling rgba uchar4 (after kernel)"); // facultatif, for debug only, remove for release
    }

/**
 * Override
 * Call periodicly by the API
 */
void Mandelbrot::animationStep()
    {
    t = variateurAnimation.varierAndGet();
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

