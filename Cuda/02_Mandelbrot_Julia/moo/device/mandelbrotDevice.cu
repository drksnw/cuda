#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "cudaTools.h"
#include "Device.h"

#include "MathTools.h"

#include "IndiceTools_GPU.h"
#include "DomaineMath_GPU.h"

#include "MandelbrotMath.h"
using namespace gpu;

// Attention : 	Choix du nom est impotant!
//		VagueDevice.cu et non Vague.cu
// 		Dans ce dernier cas, probl�me de linkage, car le nom du .cu est le meme que le nom d'un .cpp (host)
//		On a donc ajouter Device (ou n'importequoi) pour que les noms soient diff�rents!

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void mandelbrot(uchar4* ptrDevPixels, uint w, uint h, float t, DomaineMath domaineMath);
__device__ void workPixel(uchar4* ptrColorIJ, int i, int j, const DomaineMath& domaineMath, MandelbrotMath* ptrMandelbrotMath);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void mandelbrot(uchar4* ptrDevPixels, uint w, uint h, float t, DomaineMath domaineMath)
    {
    MandelbrotMath mandelbrotMath(t);

    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();
    const int WH = w * h;

    int s = TID;
    int i;
    int j;

    while (s < WH)
	{
	IndiceTools::toIJ(s, w, &i, &j);

	workPixel(&ptrDevPixels[s], i, j, domaineMath, &mandelbrotMath);

	s += NB_THREAD;
	}
    }

__device__ void workPixel(uchar4* ptrColorIJ, int i, int j, const DomaineMath& domaineMath, MandelbrotMath* ptrMandelbrotMath)
    {
    double x;
    double y;

    domaineMath.toXY(i, j, &x, &y);

    ptrMandelbrotMath->colorXY(ptrColorIJ, x, y);
    }


/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
