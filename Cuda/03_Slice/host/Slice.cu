#include "hip/hip_runtime.h"
#include <iostream>

#include "Device.h"
#include "Slice.h"
#include <cmath>

using std::cout;
using std::endl;


extern __global__ void slice(float* ptrDevPi, int nbSlice);

#define PI 3.14159265358979323846264338327950288419716939937510


Slice::Slice(const Grid& grid, int nbSlice, float tolerance)
    {
    // Grid
	{
	this->dg = grid.dg;
	this->db = grid.db;
	}

    this->sizeOctet = db.x * db.y * db.z * sizeof(float); // octet
    this->nbSlice = nbSlice;
    this->tolerance = tolerance;
    this->pi = 0;

    // MM
	{

	// MM (malloc Device)
	    {
	    Device::malloc(&ptrDevPi, sizeof(float));
	    Device::memclear(ptrDevPi, sizeof(float));
	    }

	Device::lastCudaError("AddVector MM (end allocation)"); // temp debug, facultatif
	}

    }

Slice::~Slice(void)
    {
    //MM (device free)
	{
	Device::free(ptrDevPi);

	Device::lastCudaError("AddVector MM (end deallocation)"); // temp debug, facultatif
	}
    }


float Slice::run()
    {
    Device::lastCudaError("addVecteur (before)"); // temp debug
    slice<<<dg, db, sizeOctet>>>(ptrDevPi, nbSlice); // assynchrone
    Device::lastCudaError("addVecteur (after)"); // temp debug

    Device::memcpyDToH(&pi, ptrDevPi, sizeof(float)); // barriere synchronisation implicite

    return pi;
    }

void Slice::display()
    {
    cout << "///////////////////////" << endl;
    cout << "/////TP SLICE - PI/////" << endl;
    cout << "///////////////////////" << endl;

    cout << "Exact Pi : \t" << PI << endl;
    cout << "Estimation : \t" << this->pi << endl;

    cout << "///////////////////////" << endl;
    cout << "//////////END//////////" << endl;
    cout << "///////////////////////" << endl;
    }

bool Slice::check()
    {
    return std::fabs(this->pi - PI) < this->tolerance;
}
