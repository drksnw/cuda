#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "Indice1D.h"
#include "cudaTools.h"
#include "reductionADD.h"

#include <stdio.h>


__global__ void slice(float* ptrDevPi, int nbSlice);

__device__ void reducIntraThread(float* tab_SM, int nbSlice);
__device__ float fpi(float x);



/**
 * output : void required !!
 */
__global__ void slice(float* ptrDevPi, int nbSlice)
    {
    extern __shared__ float tab_SM[]; //size defined in calling of kernel

    reducIntraThread(tab_SM, nbSlice);
    __syncthreads();
    reductionADD<float>(tab_SM, ptrDevPi);

    }

__device__ void reducIntraThread(float* tab_SM, int nbSlice)
    {
    const int NB_THREAD=Indice2D::nbThread();
    const int TID=Indice2D::tid();
    const int TID_LOCAL=Indice2D::tidLocal();
    const float DX = 1.f / (float) nbSlice;

    int s = TID;
    float tmp = 0;
    while(s < nbSlice)
	{
	tmp += fpi(s * DX);
	s += NB_THREAD;
	}

    tab_SM[TID_LOCAL] = tmp * DX;
    }


__device__ float fpi(float x)
    {
    return 4.f / (1.f + x * x);
}
